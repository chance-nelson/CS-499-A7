#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include <omp.h>


//#define N 100 


using namespace std;


void warmUpGPU();
__global__ void vectorAdd(unsigned int * A, unsigned int * B, unsigned int * C);


int main(int argc, char *argv[]) {
    printf("RUNNING FOR N=%d\n", N);
    
    double t_start;

    warmUpGPU();
	
	unsigned int * A;
	unsigned int * B;
	unsigned int * C;
	unsigned int * C_CPU;

	A     = (unsigned int *) malloc(sizeof(unsigned int)*N);
	B     = (unsigned int *) malloc(sizeof(unsigned int)*N);
	C     = (unsigned int *) malloc(sizeof(unsigned int)*N);
	C_CPU = (unsigned int *) malloc(sizeof(unsigned int)*N);


	printf("Size of A+B+C (GiB): %f\n",(sizeof(unsigned int)*N*3.0)/(1024.0*1024.0*1024.0));
	
	
    //init:
	int i = 0;
	for(i = 0; i < N; i++) {
		A[i]     = i;
		B[i]     = i;
		C[i]     = 0;
		C_CPU[i] = 0;
	}

	//CPU version:
	
    t_start = omp_get_wtime();
	
    for(int i = 0; i < N; i++) {
		C_CPU[i] = A[i] + B[i];
	}
	
    printf("CPU TIME: %lf\n", omp_get_wtime() - t_start);
	
	//CUDA error code:
	hipError_t errCode = hipSuccess;

	unsigned int * dev_A;
	unsigned int * dev_B;
	unsigned int * dev_C;

	//allocate on the device: A, B, C
	errCode = hipMalloc((unsigned int**)&dev_A, sizeof(unsigned int)*N);	
	if(errCode != hipSuccess) {
	    cout << "\nError: A error with code " << errCode << endl; 
	}

	errCode = hipMalloc((unsigned int**)&dev_B, sizeof(unsigned int)*N);	
	if(errCode != hipSuccess) {
	    cout << "\nError: B error with code " << errCode << endl; 
	}

	errCode = hipMalloc((unsigned int**)&dev_C, sizeof(unsigned int)*N);	
	if(errCode != hipSuccess) {
	    cout << "\nError: C error with code " << errCode << endl; 
	}

    t_start = omp_get_wtime();
	
    //copy A to device
	errCode = hipMemcpy(dev_A, A, sizeof(unsigned int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	    cout << "\nError: A memcpy error with code " << errCode << endl; 
	}	
	
	//copy B to device
	errCode = hipMemcpy(dev_B, B, sizeof(unsigned int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	    cout << "\nError: A memcpy error with code " << errCode << endl; 
	}

	//copy C to device (initialized to 0)
	errCode = hipMemcpy(dev_C, C, sizeof(unsigned int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	    cout << "\nError: A memcpy error with code " << errCode << endl; 
	}

    printf("CPU->GPU COPY TIME: %lf\n", omp_get_wtime() - t_start);

	//execute kernel
	const unsigned int totalBlocks=ceil(N*1.0/1024.0);
	
    printf("total blocks: %d\n",totalBlocks);

    t_start = omp_get_wtime();    
    
    vectorAdd<<<totalBlocks,1024>>>(dev_A, dev_B, dev_C);

    hipDeviceSynchronize();

    printf("GPU RUN TIME: %lf\n", omp_get_wtime() - t_start);

	if(errCode != hipSuccess) {
		cout<<"Error after kernel launch "<<errCode<<endl;
	}


    t_start = omp_get_wtime();

	//copy data from device to host 
	errCode=hipMemcpy( C, dev_C, sizeof(unsigned int)*N, hipMemcpyDeviceToHost);
	if(errCode != hipSuccess) {
	    cout << "\nError: getting C result form GPU error with code " << errCode << endl; 
	}

    printf("GPU->CPU COPY TIME: %lf\n", omp_get_wtime() - t_start);
	
    return 0;
}


__global__ void vectorAdd(unsigned int * A, unsigned int * B, unsigned int * C) {
    unsigned int tid=threadIdx.x+ (blockIdx.x*blockDim.x); 

    if (tid>=N){
        return;
    }

    C[tid]=A[tid]+B[tid];

    return;
}


__global__ void warmup(unsigned int * tmp) {
    if (threadIdx.x==0) *tmp = 555;

    return;
}


void warmUpGPU() {
    printf("Warming up GPU for time trialing...\n");	
    
    unsigned int *dev_tmp;
    unsigned int *tmp;
    
    tmp  = (unsigned int*) malloc(sizeof(unsigned int));
    *tmp = 0;
    
    hipError_t errCode = hipSuccess;
    
    errCode = hipMalloc((unsigned int**) &dev_tmp, sizeof(unsigned int));	
    
    if(errCode != hipSuccess) {
        cout << "\nError: dev_tmp error with code " << errCode << endl; 
    }

    warmup<<<1,256>>>(dev_tmp);

    //copy data from device to host 
    errCode=hipMemcpy(tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);
    if(errCode != hipSuccess) {
        cout << "\nError: getting tmp result form GPU error with code " << errCode << endl; 
    }

    hipDeviceSynchronize();

    printf("\ntmp (changed to 555 on GPU): %d", *tmp);

    hipFree(dev_tmp);

    return;
}
